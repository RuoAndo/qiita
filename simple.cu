#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

void validateResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("Arrays do not match!\n");
            printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
            break;
        }
    }

    if (match) printf("Arrays match.\n\n");

    return;
}

void genData(float *ip, int size)
{
    time_t t;
    srand((unsigned) time(&t));

    for (int i = 0; i < size; i++)
    {
        ip[i] = (float)( rand() & 0xFF ) / 10.0f;
    }

    return;
}

void sumArraysOnHost(float *A, float *B, float *C, const int N)
{
    for (int idx = 0; idx < N; idx++)
    {
        C[idx] = A[idx] + B[idx];
    }
}
__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) C[i] = A[i] + B[i];
}

int main(int argc, char **argv)
{
    struct timespec startTime, endTime, sleepTime;

    printf("%s Starting...\n", argv[0]);

    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    hipSetDevice(dev);

    int nElem = 1 << 28;
    printf("Vector size %d\n", nElem);

    size_t nBytes = nElem * sizeof(float);

    float *host_A, *host_B, *hostRef, *gpuRef;
    host_A     = (float *)malloc(nBytes);
    host_B     = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef  = (float *)malloc(nBytes);

    genData(host_A, nElem);
    genData(host_B, nElem);

    memset(hostRef, 0, nBytes);
    memset(gpuRef,  0, nBytes);

    clock_gettime(CLOCK_REALTIME, &startTime);
    sleepTime.tv_sec = 0;
    sleepTime.tv_nsec = 123;

    sumArraysOnHost(host_A, host_B, hostRef, nElem);

    clock_gettime(CLOCK_REALTIME, &endTime);
    if (endTime.tv_nsec < startTime.tv_nsec) {
	printf("%ld.%09ld", endTime.tv_sec - startTime.tv_sec - 1
	       ,endTime.tv_nsec + 1000000000 - startTime.tv_nsec);
    } else {
	printf("%ld.%09ld", endTime.tv_sec - startTime.tv_sec
	       ,endTime.tv_nsec - startTime.tv_nsec);
    }
    printf(" sec\n");

    float *device_A, *device_B, *device_C;
    hipMalloc((float**)&device_A, nBytes);
    hipMalloc((float**)&device_B, nBytes);
    hipMalloc((float**)&device_C, nBytes);

    hipMemcpy(device_A, host_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(device_B, host_B, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(device_C, gpuRef, nBytes, hipMemcpyHostToDevice);

    int iLen = 1024;
    dim3 block (iLen);
    dim3 grid  ((nElem + block.x - 1) / block.x);

    clock_gettime(CLOCK_REALTIME, &startTime);
    sleepTime.tv_sec = 0;
    sleepTime.tv_nsec = 123;

    sumArraysOnGPU<<<grid, block>>>(device_A, device_B, device_C, nElem);
    hipDeviceSynchronize();
    hipGetLastError() ;

    clock_gettime(CLOCK_REALTIME, &endTime);
    if (endTime.tv_nsec < startTime.tv_nsec) {
	printf("%ld.%09ld", endTime.tv_sec - startTime.tv_sec - 1
	       ,endTime.tv_nsec + 1000000000 - startTime.tv_nsec);
    } else {
	printf("%ld.%09ld", endTime.tv_sec - startTime.tv_sec
	       ,endTime.tv_nsec - startTime.tv_nsec);
    }
    printf(" sec\n");

    hipMemcpy(gpuRef, device_C, nBytes, hipMemcpyDeviceToHost);

    validateResult(hostRef, gpuRef, nElem);

    hipFree(device_A);
    hipFree(device_B);
    hipFree(device_C);

    free(host_A);
    free(host_B);
    free(hostRef);
    free(gpuRef);

    return(0);
}
